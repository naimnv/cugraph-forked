#include "hip/hip_runtime.h"
#include "prims/kv_store.cuh"
#include "utilities/collect_comm.cuh"

#include <cugraph/detail/collect_comm_wrapper.hpp>
#include <cugraph/detail/decompress_edge_partition.cuh>
#include <cugraph/detail/shuffle_wrappers.hpp>
#include <cugraph/utilities/mask_utils.cuh>
#include <cugraph/utilities/misc_utils.cuh>

#include <raft/core/handle.hpp>

#include <unordered_map>

namespace cugraph {

namespace detail {

template <typename TupleType, std::size_t... Is>
constexpr TupleType invalid_of_thrust_tuple_of_integral(std::index_sequence<Is...>)
{
  return thrust::make_tuple(
    cugraph::invalid_idx<typename thrust::tuple_element<Is, TupleType>::type>::value...);
}
}  // namespace detail

template <typename TupleType>
constexpr TupleType invalid_of_thrust_tuple_of_integral()
{
  return detail::invalid_of_thrust_tuple_of_integral<TupleType>(
    std::make_index_sequence<thrust::tuple_size<TupleType>::value>());
}

template <typename edge_type_t, typename edge_id_t, typename value_t>
struct edge_type_and_id_search_container_t {
  using edge_type_type = edge_type_t;
  using edge_id_type   = edge_id_t;
  using value_type     = value_t;
  using container_t =
    std::unordered_map<edge_type_t,
                       cugraph::kv_store_t<edge_id_t, value_t, false /*use_binary_search*/>>;

  static_assert(std::is_arithmetic_v<edge_type_t>);
  static_assert(std::is_arithmetic_v<edge_id_t>);
  static_assert(is_arithmetic_or_thrust_tuple_of_arithmetic<value_t>::value);
  edge_type_and_id_search_container_t() = delete;

  edge_type_and_id_search_container_t(const edge_type_and_id_search_container_t&) = delete;

  edge_type_and_id_search_container_t& operator=(const edge_type_and_id_search_container_t&) =
    delete;

  container_t edge_type_to_kv_store;
};

template <typename GraphViewType,
          typename EdgeIdInputWrapper,
          typename EdgeTypeInputWrapper,
          typename EdgeTypeAndIdToSrcDstMapType>
std::tuple<rmm::device_uvector<typename GraphViewType::vertex_type>,
           rmm::device_uvector<typename GraphViewType::vertex_type>>
lookup(raft::handle_t const& handle,
       GraphViewType const& graph_view,
       EdgeIdInputWrapper edge_id_view,
       EdgeTypeInputWrapper edge_type_view,
       EdgeTypeAndIdToSrcDstMapType const& search_container,
       raft::device_span<typename EdgeIdInputWrapper::value_type const> edge_ids_to_lookup,
       typename EdgeTypeInputWrapper::value_type edge_type_to_lookup)
{
  using vertex_t = typename GraphViewType::vertex_type;

  using edge_t      = typename GraphViewType::edge_type;
  using edge_id_t   = typename EdgeIdInputWrapper::value_type;
  using edge_type_t = typename EdgeTypeInputWrapper::value_type;
  using value_t     = typename EdgeTypeAndIdToSrcDstMapType::value_type;
  using store_t     = typename EdgeTypeAndIdToSrcDstMapType::container_t::mapped_type;

  static_assert(std::is_same_v<value_t, thrust::tuple<vertex_t, vertex_t>>);

  static_assert(std::is_integral_v<edge_id_t>);
  static_assert(std::is_integral_v<edge_type_t>);
  static_assert(is_arithmetic_or_thrust_tuple_of_arithmetic<value_t>::value);

  static_assert(std::is_same_v<typename EdgeTypeAndIdToSrcDstMapType::edge_id_type, edge_id_t>,
                "edge_id_t must match EdgeTypeAndIdToSrcDstMapType::edge_id_type");
  static_assert(std::is_same_v<typename EdgeTypeAndIdToSrcDstMapType::edge_type_type, edge_type_t>,
                "edge_type_t must match EdgeTypeAndIdToSrcDstMapType::edge_type_type ");

  rmm::device_uvector<vertex_t> output_srcs(edge_ids_to_lookup.size(), handle.get_stream());
  rmm::device_uvector<vertex_t> output_dsts(edge_ids_to_lookup.size(), handle.get_stream());

  auto constexpr invalid_vertex_id = cugraph::invalid_vertex_id<vertex_t>::value;
  thrust::fill(
    handle.get_thrust_policy(), output_srcs.begin(), output_srcs.end(), invalid_vertex_id);
  thrust::fill(
    handle.get_thrust_policy(), output_dsts.begin(), output_dsts.end(), invalid_vertex_id);

  const store_t* kv_store_object{nullptr};

  auto itr = search_container.edge_type_to_kv_store.find(edge_type_to_lookup);
  if (itr != search_container.edge_type_to_kv_store.end()) {
    assert(edge_type_to_lookup == itr->first);
    kv_store_object = &(itr->second);
  } else {
    return std::make_tuple(std::move(output_srcs), std::move(output_dsts));
  }

  auto value_buffer = cugraph::allocate_dataframe_buffer<value_t>(0, handle.get_stream());

  if (GraphViewType::is_multi_gpu) {
    auto& comm                 = handle.get_comms();
    auto const comm_size       = comm.get_size();
    auto& major_comm           = handle.get_subcomm(cugraph::partition_manager::major_comm_name());
    auto const major_comm_size = major_comm.get_size();
    auto& minor_comm           = handle.get_subcomm(cugraph::partition_manager::minor_comm_name());
    auto const minor_comm_size = minor_comm.get_size();

    auto key_func = cugraph::detail::compute_gpu_id_from_ext_vertex_t<edge_id_t>{
      comm_size, major_comm_size, minor_comm_size};

    value_buffer = cugraph::collect_values_for_keys(handle,
                                                    kv_store_object->view(),
                                                    edge_ids_to_lookup.begin(),
                                                    edge_ids_to_lookup.end(),
                                                    key_func);

  } else {
    cugraph::resize_dataframe_buffer(value_buffer, edge_ids_to_lookup.size(), handle.get_stream());

    kv_store_object->view().find(edge_ids_to_lookup.begin(),
                                 edge_ids_to_lookup.end(),
                                 cugraph::get_dataframe_buffer_begin(value_buffer),
                                 handle.get_stream());
  }

  thrust::copy(
    handle.get_thrust_policy(),
    cugraph::get_dataframe_buffer_begin(value_buffer),
    cugraph::get_dataframe_buffer_end(value_buffer),
    thrust::make_zip_iterator(thrust::make_tuple(output_srcs.begin(), output_dsts.begin())));

  return std::make_tuple(std::move(output_srcs), std::move(output_dsts));
}

template <typename GraphViewType,
          typename EdgeIdInputWrapper,
          typename EdgeTypeInputWrapper,
          typename EdgeTypeAndIdToSrcDstMapType>
EdgeTypeAndIdToSrcDstMapType create_edge_id_lookup_map(raft::handle_t const& handle,
                                                       GraphViewType const& graph_view,
                                                       EdgeIdInputWrapper edge_id_view,
                                                       EdgeTypeInputWrapper edge_type_view)
{
  static_assert(!std::is_same_v<typename EdgeIdInputWrapper::value_type, thrust::nullopt_t>,
                "Can not create edge id lookup table without edge ids");

  using vertex_t    = typename GraphViewType::vertex_type;
  using edge_t      = typename GraphViewType::edge_type;
  using edge_type_t = typename EdgeTypeInputWrapper::value_type;
  using edge_id_t   = typename EdgeIdInputWrapper::value_type;
  using value_t     = typename EdgeTypeAndIdToSrcDstMapType::value_type;
  using container_t = typename EdgeTypeAndIdToSrcDstMapType::container_t;
  using store_t     = typename EdgeTypeAndIdToSrcDstMapType::container_t::mapped_type;

  constexpr bool multi_gpu = GraphViewType::is_multi_gpu;

  static_assert(std::is_integral_v<edge_type_t>);
  static_assert(std::is_integral_v<edge_id_t>);
  static_assert(is_arithmetic_or_thrust_tuple_of_arithmetic<value_t>::value);

  static_assert(std::is_same_v<value_t, thrust::tuple<vertex_t, vertex_t>>);

  static_assert(std::is_same_v<typename EdgeTypeAndIdToSrcDstMapType::edge_type_type, edge_type_t>,
                "edge_type_t must match with EdgeTypeAndIdToSrcDstMapType::edge_type_type");

  static_assert(std::is_same_v<typename EdgeTypeAndIdToSrcDstMapType::edge_id_type, edge_id_t>,
                "edge_id_t must match with typename EdgeTypeAndIdToSrcDstMapType::edge_id_type");

  std::vector<edge_type_t> h_types_to_this_gpu{};
  std::vector<edge_t> h_freq_of_types_to_this_gpu{};
  std::unordered_map<edge_type_t, edge_t> edge_type_to_count_map{};

  for (size_t i = 0; i < graph_view.number_of_local_edge_partitions(); ++i) {
    //
    //  Copy edge ids and types
    //

    auto number_of_edges_partition = graph_view.local_edge_partition_view(i).number_of_edges();
    auto number_of_active_edges_partition = number_of_edges_partition;

    if (graph_view.has_edge_mask()) {
      number_of_active_edges_partition = detail::count_set_bits(
        handle, (*(graph_view.edge_mask_view())).value_firsts()[i], number_of_edges_partition);
    }

    [[maybe_unused]] auto edgelist_ids = std::make_optional<rmm::device_uvector<edge_t>>(
      number_of_active_edges_partition, handle.get_stream());

    auto edgelist_types = std::make_optional<rmm::device_uvector<edge_t>>(
      number_of_active_edges_partition, handle.get_stream());

    auto edge_partition_mask_view =
      graph_view.has_edge_mask()
        ? std::make_optional<
            detail::edge_partition_edge_property_device_view_t<edge_t, uint32_t const*, bool>>(
            *(graph_view.edge_mask_view()), i)
        : std::nullopt;

    if (edge_partition_mask_view) {
      if constexpr (multi_gpu) {
        detail::copy_if_mask_set(handle,
                                 edge_id_view.value_firsts()[i],
                                 edge_id_view.value_firsts()[i] + number_of_edges_partition,
                                 (*edge_partition_mask_view).value_first(),
                                 (*edgelist_ids).begin());
      }
      detail::copy_if_mask_set(handle,
                               edge_type_view.value_firsts()[i],
                               edge_type_view.value_firsts()[i] + number_of_edges_partition,
                               (*edge_partition_mask_view).value_first(),
                               (*edgelist_types).begin());

    } else {
      if constexpr (multi_gpu) {
        thrust::copy(handle.get_thrust_policy(),
                     edge_id_view.value_firsts()[i],
                     edge_id_view.value_firsts()[i] + number_of_edges_partition,
                     (*edgelist_ids).begin());
      }
      thrust::copy(handle.get_thrust_policy(),
                   edge_type_view.value_firsts()[i],
                   edge_type_view.value_firsts()[i] + number_of_edges_partition,
                   (*edgelist_types).begin());
    }

    std::vector<int> h_unique_gpu_ids{};
    std::vector<edge_t> h_gpu_offsets{};

    if constexpr (multi_gpu) {
      //
      // Count number of edge ids mapped to each GPU
      //

      auto& comm           = handle.get_comms();
      auto const comm_size = comm.get_size();
      auto& major_comm     = handle.get_subcomm(cugraph::partition_manager::major_comm_name());
      auto const major_comm_size = major_comm.get_size();
      auto& minor_comm = handle.get_subcomm(cugraph::partition_manager::minor_comm_name());
      auto const minor_comm_size = minor_comm.get_size();

      // destination gpu id from edge id. NB: edgelist_ids will holds gpu ids after the following
      // thrust::transform
      thrust::transform(handle.get_thrust_policy(),
                        (*edgelist_ids).begin(),
                        (*edgelist_ids).end(),
                        (*edgelist_ids).begin(),
                        [key_func = cugraph::detail::compute_gpu_id_from_ext_vertex_t<edge_t>{
                           comm_size, major_comm_size, minor_comm_size}] __device__(auto eid) {
                          return key_func(eid);
                        });

      thrust::sort(handle.get_thrust_policy(),
                   thrust::make_zip_iterator(
                     thrust::make_tuple((*edgelist_ids).begin(), (*edgelist_types).begin())),
                   thrust::make_zip_iterator(
                     thrust::make_tuple((*edgelist_ids).end(), (*edgelist_types).end())));

      auto nr_unique_gpu_ids =
        thrust::count_if(handle.get_thrust_policy(),
                         thrust::make_counting_iterator(size_t{0}),
                         thrust::make_counting_iterator((*edgelist_ids).size()),
                         detail::is_first_in_run_t<edge_t const*>{(*edgelist_ids).data()});

      rmm::device_uvector<int> unique_gpu_ids(nr_unique_gpu_ids, handle.get_stream());
      rmm::device_uvector<edge_t> gpu_offsets(nr_unique_gpu_ids + 1, handle.get_stream());

      thrust::fill(handle.get_thrust_policy(), gpu_offsets.begin(), gpu_offsets.end(), edge_t{0});

      thrust::reduce_by_key(handle.get_thrust_policy(),
                            (*edgelist_ids).begin(),
                            (*edgelist_ids).end(),
                            thrust::make_constant_iterator(size_t{1}),
                            unique_gpu_ids.begin(),
                            gpu_offsets.begin());

      thrust::exclusive_scan(handle.get_thrust_policy(),
                             gpu_offsets.begin(),
                             gpu_offsets.end(),
                             gpu_offsets.begin(),
                             size_t{0});

      h_unique_gpu_ids.resize(unique_gpu_ids.size());
      h_gpu_offsets.resize(gpu_offsets.size());
      raft::update_host(
        h_unique_gpu_ids.data(), unique_gpu_ids.data(), unique_gpu_ids.size(), handle.get_stream());
      raft::update_host(
        h_gpu_offsets.data(), gpu_offsets.data(), gpu_offsets.size(), handle.get_stream());

    } else {
      thrust::sort(handle.get_thrust_policy(), (*edgelist_types).begin(), (*edgelist_types).end());

      h_unique_gpu_ids.resize(size_t{1});
      h_unique_gpu_ids.push_back(0);

      h_gpu_offsets.resize(h_unique_gpu_ids.size() + 1);
      h_gpu_offsets.push_back(0);
      h_gpu_offsets.push_back((*edgelist_types).size());
    }
    //
    // For edge ids mapped to each gpu, count number of unique types and elements per type.
    // hipcub::DeviceSegmentedReduce(ByKey) ???
    //

    [[maybe_unused]] std::vector<int> h_gpu_ids_partition{};
    std::vector<edge_type_t> h_types_partition{};
    std::vector<edge_t> h_type_freqs_partition{};

    rmm::device_uvector<edge_type_t> unique_types_segment(0, handle.get_stream());
    rmm::device_uvector<edge_t> type_freqs_segment(0, handle.get_stream());

    for (size_t j = 0; j < h_unique_gpu_ids.size(); ++j) {
      auto gpu_id = h_unique_gpu_ids[i];

      auto nr_uniqe_types_segment = thrust::count_if(
        handle.get_thrust_policy(),
        thrust::make_counting_iterator(size_t{0}),
        thrust::make_counting_iterator(
          static_cast<size_t>(h_gpu_offsets[i + 1] - h_gpu_offsets[i])),
        detail::is_first_in_run_t<edge_t const*>{(*edgelist_types).data() + h_gpu_offsets[i]});

      unique_types_segment.resize(nr_uniqe_types_segment, handle.get_stream());
      type_freqs_segment.resize(nr_uniqe_types_segment, handle.get_stream());

      thrust::reduce_by_key(handle.get_thrust_policy(),
                            (*edgelist_types).begin() + h_gpu_offsets[i],
                            (*edgelist_types).begin() + h_gpu_offsets[i + 1],
                            thrust::make_constant_iterator(size_t{1}),
                            unique_types_segment.begin(),
                            type_freqs_segment.begin());

      std::vector<edge_type_t> h_unique_types_segment(nr_uniqe_types_segment);
      std::vector<edge_t> h_type_freqs_segment(nr_uniqe_types_segment);

      raft::update_host(h_unique_types_segment.data(),
                        unique_types_segment.data(),
                        unique_types_segment.size(),
                        handle.get_stream());
      raft::update_host(h_type_freqs_segment.data(),
                        type_freqs_segment.data(),
                        type_freqs_segment.size(),
                        handle.get_stream());

      if constexpr (multi_gpu) {
        h_gpu_ids_partition.insert(h_gpu_ids_partition.end(), nr_uniqe_types_segment, gpu_id);
      }

      h_types_partition.insert(
        h_types_partition.end(), h_unique_types_segment.begin(), h_unique_types_segment.end());
      h_type_freqs_partition.insert(
        h_type_freqs_partition.end(), h_type_freqs_segment.begin(), h_type_freqs_segment.end());
    }

    [[maybe_unused]] rmm::device_uvector<int> gpu_ids_partition(h_gpu_ids_partition.size(),
                                                                handle.get_stream());
    rmm::device_uvector<edge_type_t> types_partition(h_types_partition.size(), handle.get_stream());
    rmm::device_uvector<edge_t> type_freqs_partition(h_type_freqs_partition.size(),
                                                     handle.get_stream());

    if constexpr (multi_gpu) {
      raft::update_device(gpu_ids_partition.data(),
                          h_gpu_ids_partition.data(),
                          h_gpu_ids_partition.size(),
                          handle.get_stream());
    }
    raft::update_device(types_partition.data(),
                        h_types_partition.data(),
                        h_types_partition.size(),
                        handle.get_stream());
    raft::update_device(type_freqs_partition.data(),
                        h_type_freqs_partition.data(),
                        h_type_freqs_partition.size(),
                        handle.get_stream());

    if constexpr (multi_gpu) {
      // Shuffle to the proper GPUs
      std::forward_as_tuple(std::tie(gpu_ids_partition, types_partition, type_freqs_partition),
                            std::ignore) =
        cugraph::groupby_gpu_id_and_shuffle_values(
          handle.get_comms(),
          thrust::make_zip_iterator(thrust::make_tuple(
            gpu_ids_partition.begin(), types_partition.begin(), type_freqs_partition.begin())),
          thrust::make_zip_iterator(thrust::make_tuple(
            gpu_ids_partition.end(), types_partition.end(), type_freqs_partition.end())),
          [] __device__(auto val) { return thrust::get<0>(val); },
          handle.get_stream());

      thrust::for_each(
        handle.get_thrust_policy(),
        gpu_ids_partition.begin(),
        gpu_ids_partition.end(),
        [rank = handle.get_comms().get_rank()] __device__(auto val) { assert(val == rank); });
    }

    thrust::sort_by_key(handle.get_thrust_policy(),
                        types_partition.begin(),
                        types_partition.end(),
                        type_freqs_partition.begin());

    auto nr_uniqe_types_partition =
      thrust::count_if(handle.get_thrust_policy(),
                       thrust::make_counting_iterator(size_t{0}),
                       thrust::make_counting_iterator(types_partition.size()),
                       detail::is_first_in_run_t<edge_type_t const*>{types_partition.data()});

    rmm::device_uvector<edge_type_t> unique_types_partition(nr_uniqe_types_partition,
                                                            handle.get_stream());
    rmm::device_uvector<edge_t> unique_type_freqs_partition(nr_uniqe_types_partition,
                                                            handle.get_stream());

    thrust::reduce_by_key(handle.get_thrust_policy(),
                          types_partition.begin(),
                          types_partition.end(),
                          type_freqs_partition.begin(),
                          unique_types_partition.begin(),
                          unique_type_freqs_partition.begin());

    std::vector<edge_type_t> h_unique_types_partition(nr_uniqe_types_partition);
    std::vector<edge_t> h_unique_type_freqs_partition(nr_uniqe_types_partition);

    raft::update_host(h_unique_types_partition.data(),
                      unique_types_partition.data(),
                      unique_types_partition.size(),
                      handle.get_stream());

    raft::update_host(h_unique_type_freqs_partition.data(),
                      unique_type_freqs_partition.data(),
                      unique_type_freqs_partition.size(),
                      handle.get_stream());

    for (edge_type_t j = 0; j < nr_uniqe_types_partition; j++) {
      auto typ  = h_unique_types_partition[i];
      auto freq = h_unique_type_freqs_partition[i];
      if (edge_type_to_count_map.find(typ) != edge_type_to_count_map.end()) {
        edge_type_to_count_map[typ] += freq;
      } else {
        edge_type_to_count_map[typ] = freq;
      }
    }

    h_types_to_this_gpu.insert(
      h_types_to_this_gpu.end(), h_unique_types_partition.begin(), h_unique_types_partition.end());

    h_freq_of_types_to_this_gpu.insert(h_freq_of_types_to_this_gpu.end(),
                                       h_unique_type_freqs_partition.begin(),
                                       h_unique_type_freqs_partition.end());
  }

  //
  // Find global unique types and their frequencies
  //
  rmm::device_uvector<edge_type_t> types_to_this_gpu(h_types_to_this_gpu.size(),
                                                     handle.get_stream());

  rmm::device_uvector<edge_t> freq_of_types_to_this_gpu(h_freq_of_types_to_this_gpu.size(),
                                                        handle.get_stream());

  raft::update_device(types_to_this_gpu.data(),
                      h_types_to_this_gpu.data(),
                      h_types_to_this_gpu.size(),
                      handle.get_stream());

  raft::update_device(freq_of_types_to_this_gpu.data(),
                      h_freq_of_types_to_this_gpu.data(),
                      h_freq_of_types_to_this_gpu.size(),
                      handle.get_stream());

  thrust::sort_by_key(handle.get_thrust_policy(),
                      types_to_this_gpu.begin(),
                      types_to_this_gpu.end(),
                      freq_of_types_to_this_gpu.begin());

  auto nr_unique_types_this_gpu =
    thrust::count_if(handle.get_thrust_policy(),
                     thrust::make_counting_iterator(size_t{0}),
                     thrust::make_counting_iterator(types_to_this_gpu.size()),
                     detail::is_first_in_run_t<edge_type_t const*>{types_to_this_gpu.data()});

  rmm::device_uvector<edge_type_t> unique_types_to_this_gpu(nr_unique_types_this_gpu,
                                                            handle.get_stream());
  rmm::device_uvector<edge_t> freq_of_unique_types_this_gpu(nr_unique_types_this_gpu,
                                                            handle.get_stream());

  thrust::reduce_by_key(handle.get_thrust_policy(),
                        types_to_this_gpu.begin(),
                        types_to_this_gpu.end(),
                        freq_of_types_to_this_gpu.begin(),
                        unique_types_to_this_gpu.begin(),
                        freq_of_unique_types_this_gpu.begin());

  rmm::device_uvector<int> global_unique_types(nr_unique_types_this_gpu, handle.get_stream());

  thrust::transform(handle.get_thrust_policy(),
                    unique_types_to_this_gpu.begin(),
                    unique_types_to_this_gpu.end(),
                    global_unique_types.begin(),
                    [] __device__(auto val) { return int{val}; });

  auto nr_unique_types_global = nr_unique_types_this_gpu;

  if constexpr (multi_gpu) {
    global_unique_types = cugraph::detail::shuffle_ext_vertices_to_local_gpu_by_vertex_partitioning(
      handle, std::move(global_unique_types));

    thrust::sort(
      handle.get_thrust_policy(), global_unique_types.begin(), global_unique_types.end());

    auto nr_unique_elements = static_cast<size_t>(thrust::distance(
      global_unique_types.begin(),
      thrust::unique(
        handle.get_thrust_policy(), global_unique_types.begin(), global_unique_types.end())));

    global_unique_types.resize(nr_unique_elements, handle.get_stream());

    nr_unique_types_global = host_scalar_allreduce(
      handle.get_comms(), nr_unique_elements, raft::comms::op_t::SUM, handle.get_stream());

    global_unique_types = detail::device_allgatherv(
      handle,
      handle.get_comms(),
      raft::device_span<int const>{global_unique_types.data(), global_unique_types.size()});

    assert(global_unique_types.size() == nr_unique_types_global);
  }

  std::vector<int> h_unique_types_global(nr_unique_types_global);

  raft::update_host(h_unique_types_global.data(),
                    global_unique_types.data(),
                    global_unique_types.size(),
                    handle.get_stream());

  //
  // Create search container with appropriate capacity
  //

  container_t edge_type_to_kv_store_umap{};
  edge_type_to_kv_store_umap.reserve(nr_unique_types_global);

  auto invalid_key   = cugraph::invalid_vertex_id<edge_id_t>::value;
  auto invalid_value = (std::is_integral<value_t>::value)
                         ? (cugraph::invalid_vertex_id<vertex_t>::value)
                         : invalid_of_thrust_tuple_of_integral<value_t>();

  for (size_t idx = 0; idx < h_unique_types_global.size(); idx++) {
    auto typ              = h_unique_types_global[idx];
    auto search_itr       = edge_type_to_count_map.find(typ);
    size_t store_capacity = (search_itr != edge_type_to_count_map.end()) ? search_itr->second : 0;

    edge_type_to_kv_store_umap.insert(
      {typ, std::move(store_t(store_capacity, invalid_key, invalid_value, handle.get_stream()))});

    assert(edge_type_to_kv_store_umap.find(typ) != edge_type_to_kv_store_umap.end());
  }

  //
  // Populate the search container
  //

  for (size_t i = 0; i < graph_view.number_of_local_edge_partitions(); ++i) {
    //
    // decompress one edge_partition at a time
    //

    auto number_of_local_edges = graph_view.local_edge_partition_view(i).number_of_edges();

    if (graph_view.has_edge_mask()) {
      number_of_local_edges = detail::count_set_bits(
        handle, (*(graph_view.edge_mask_view())).value_firsts()[i], number_of_local_edges);
    }

    rmm::device_uvector<vertex_t> edgelist_majors(number_of_local_edges, handle.get_stream());
    rmm::device_uvector<vertex_t> edgelist_minors(edgelist_majors.size(), handle.get_stream());
    auto edgelist_ids =
      std::make_optional<rmm::device_uvector<edge_t>>(edgelist_majors.size(), handle.get_stream());
    auto edgelist_types = std::make_optional<rmm::device_uvector<edge_type_t>>(
      edgelist_majors.size(), handle.get_stream());

    detail::decompress_edge_partition_to_edgelist<vertex_t, edge_t, edge_type_t, multi_gpu>(
      handle,
      edge_partition_device_view_t<vertex_t, edge_t, multi_gpu>(
        graph_view.local_edge_partition_view(i)),
      std::make_optional<
        detail::edge_partition_edge_property_device_view_t<edge_t, edge_type_t const*>>(
        edge_type_view, i),
      std::make_optional<detail::edge_partition_edge_property_device_view_t<edge_t, edge_t const*>>(
        edge_id_view, i),
      graph_view.has_edge_mask()
        ? std::make_optional<
            detail::edge_partition_edge_property_device_view_t<edge_t, uint32_t const*, bool>>(
            *(graph_view.edge_mask_view()), i)
        : std::nullopt,
      raft::device_span<vertex_t>(edgelist_majors.data(), number_of_local_edges),
      raft::device_span<vertex_t>(edgelist_minors.data(), number_of_local_edges),
      std::make_optional<raft::device_span<edge_type_t>>((*edgelist_types).data(),
                                                         number_of_local_edges),
      std::make_optional<raft::device_span<edge_t>>((*edgelist_ids).data(), number_of_local_edges),
      graph_view.local_edge_partition_segment_offsets(i));

    //
    // Shuffle to the right GPUs using edge ids as keys
    //

    if constexpr (multi_gpu) {
      auto const comm_size = handle.get_comms().get_size();
      auto& major_comm     = handle.get_subcomm(cugraph::partition_manager::major_comm_name());
      auto const major_comm_size = major_comm.get_size();
      auto& minor_comm = handle.get_subcomm(cugraph::partition_manager::minor_comm_name());
      auto const minor_comm_size = minor_comm.get_size();

      auto key_func = cugraph::detail::compute_gpu_id_from_ext_vertex_t<vertex_t>{
        comm_size, major_comm_size, minor_comm_size};

      rmm::device_uvector<size_t> d_tx_value_counts(0, handle.get_stream());

      auto itr_to_quadruple = thrust::make_zip_iterator(edgelist_majors.begin(),
                                                        edgelist_minors.begin(),
                                                        (*edgelist_ids).begin(),
                                                        (*edgelist_types).begin());

      d_tx_value_counts = cugraph::groupby_and_count(
        itr_to_quadruple,
        itr_to_quadruple + edgelist_majors.size(),
        [key_func] __device__(auto val) { return key_func(thrust::get<2>(val)); },
        comm_size,
        std::numeric_limits<edge_id_t>::max(), /* mem_frugal_threshold */
        handle.get_stream());

      std::vector<size_t> h_tx_value_counts(d_tx_value_counts.size());
      raft::update_host(h_tx_value_counts.data(),
                        d_tx_value_counts.data(),
                        d_tx_value_counts.size(),
                        handle.get_stream());

      std::forward_as_tuple(
        std::tie(edgelist_majors, edgelist_minors, (*edgelist_ids), (*edgelist_types)),
        std::ignore) =
        shuffle_values(
          handle.get_comms(), itr_to_quadruple, h_tx_value_counts, handle.get_stream());
    }

    //
    // Sort by edge types and insert to type specific kv_store_t object
    //

    auto itr_to_triple = thrust::make_zip_iterator(
      edgelist_majors.begin(), edgelist_minors.begin(), (*edgelist_ids).begin());

    thrust::sort_by_key(handle.get_thrust_policy(),
                        (*edgelist_types).begin(),
                        (*edgelist_types).end(),
                        itr_to_triple);

    auto nr_uniqe_edge_types_partition =
      thrust::count_if(handle.get_thrust_policy(),
                       thrust::make_counting_iterator(size_t{0}),
                       thrust::make_counting_iterator((*edgelist_types).size()),
                       detail::is_first_in_run_t<edge_type_t const*>{(*edgelist_types).data()});

    rmm::device_uvector<edge_type_t> unique_types(nr_uniqe_edge_types_partition,
                                                  handle.get_stream());
    rmm::device_uvector<edge_t> type_offsets(nr_uniqe_edge_types_partition + 1,
                                             handle.get_stream());

    thrust::fill(handle.get_thrust_policy(), type_offsets.begin(), type_offsets.end(), edge_t{0});

    thrust::reduce_by_key(handle.get_thrust_policy(),
                          (*edgelist_types).begin(),
                          (*edgelist_types).end(),
                          thrust::make_constant_iterator(size_t{1}),
                          unique_types.begin(),
                          type_offsets.begin());

    thrust::exclusive_scan(handle.get_thrust_policy(),
                           type_offsets.begin(),
                           type_offsets.end(),
                           type_offsets.begin(),
                           size_t{0});

    std::vector<edge_type_t> h_unique_types(unique_types.size());
    std::vector<edge_t> h_type_offsets(type_offsets.size());

    raft::update_host(
      h_unique_types.data(), unique_types.data(), unique_types.size(), handle.get_stream());

    raft::update_host(
      h_type_offsets.data(), type_offsets.data(), type_offsets.size(), handle.get_stream());

    for (size_t idx = 0; idx < h_unique_types.size(); idx++) {
      auto typ = h_unique_types[idx];
      auto itr = edge_type_to_kv_store_umap.find(typ);
      if (itr != edge_type_to_kv_store_umap.end()) {
        assert(itr->first == typ);
        itr->second.insert((*edgelist_ids).begin() + h_type_offsets[idx],
                           (*edgelist_ids).begin() + h_type_offsets[idx + 1],
                           thrust::make_zip_iterator(
                             thrust::make_tuple(edgelist_majors.begin(), edgelist_minors.begin())) +
                             h_type_offsets[idx],
                           handle.get_stream());
      } else {
        assert(false);
      }
    }
  }

  return edge_type_and_id_search_container_t<edge_type_t, edge_id_t, value_t>{
    std::move(edge_type_to_kv_store_umap)};
}

template edge_type_and_id_search_container_t<uint8_t, int32_t, thrust::tuple<int32_t, int32_t>>
create_edge_id_lookup_map(raft::handle_t const& handle,
                          graph_view_t<int32_t, int32_t, false, true> const& graph_view,
                          edge_property_view_t<int32_t, int32_t const*> edge_id_view,
                          edge_property_view_t<int32_t, uint8_t const*> edge_type_view);

template edge_type_and_id_search_container_t<uint8_t, int64_t, thrust::tuple<int64_t, int64_t>>
create_edge_id_lookup_map(raft::handle_t const& handle,
                          graph_view_t<int64_t, int64_t, false, true> const& graph_view,
                          edge_property_view_t<int64_t, int64_t const*> edge_id_view,
                          edge_property_view_t<int64_t, uint8_t const*> edge_type_view);

template std::tuple<rmm::device_uvector<int32_t>, rmm::device_uvector<int32_t>> lookup(
  raft::handle_t const& handle,
  graph_view_t<int32_t, int32_t, false, true> const& graph_view,
  edge_property_view_t<int32_t, int32_t const*> edge_id_view,
  edge_property_view_t<int32_t, uint8_t const*> edge_type_view,
  edge_type_and_id_search_container_t<uint8_t, int32_t, thrust::tuple<int32_t, int32_t>> const&
    search_container,
  raft::device_span<int32_t const> edge_ids_to_lookup,
  uint8_t edge_type_to_lookup);

template std::tuple<rmm::device_uvector<int32_t>, rmm::device_uvector<int32_t>> lookup(
  raft::handle_t const& handle,
  graph_view_t<int32_t, int64_t, false, true> const& graph_view,
  edge_property_view_t<int64_t, int64_t const*> edge_id_view,
  edge_property_view_t<int64_t, uint8_t const*> edge_type_view,
  edge_type_and_id_search_container_t<uint8_t, int64_t, thrust::tuple<int32_t, int32_t>> const&
    search_container,
  raft::device_span<int64_t const> edge_ids_to_lookup,
  uint8_t edge_type_to_lookup);

template std::tuple<rmm::device_uvector<int64_t>, rmm::device_uvector<int64_t>> lookup(
  raft::handle_t const& handle,
  graph_view_t<int64_t, int64_t, false, true> const& graph_view,
  edge_property_view_t<int64_t, int64_t const*> edge_id_view,
  edge_property_view_t<int64_t, uint8_t const*> edge_type_view,
  edge_type_and_id_search_container_t<uint8_t, int64_t, thrust::tuple<int64_t, int64_t>> const&
    search_container,
  raft::device_span<int64_t const> edge_ids_to_lookup,
  uint8_t edge_type_to_lookup);

template std::tuple<rmm::device_uvector<int32_t>, rmm::device_uvector<int32_t>> lookup(
  raft::handle_t const& handle,
  graph_view_t<int32_t, int32_t, false, false> const& graph_view,
  edge_property_view_t<int32_t, int32_t const*> edge_id_view,
  edge_property_view_t<int32_t, uint8_t const*> edge_type_view,
  edge_type_and_id_search_container_t<uint8_t, int32_t, thrust::tuple<int32_t, int32_t>> const&
    search_container,
  raft::device_span<int32_t const> edge_ids_to_lookup,
  uint8_t edge_type_to_lookup);

template std::tuple<rmm::device_uvector<int32_t>, rmm::device_uvector<int32_t>> lookup(
  raft::handle_t const& handle,
  graph_view_t<int32_t, int64_t, false, false> const& graph_view,
  edge_property_view_t<int64_t, int64_t const*> edge_id_view,
  edge_property_view_t<int64_t, uint8_t const*> edge_type_view,
  edge_type_and_id_search_container_t<uint8_t, int64_t, thrust::tuple<int32_t, int32_t>> const&
    search_container,
  raft::device_span<int64_t const> edge_ids_to_lookup,
  uint8_t edge_type_to_lookup);

template std::tuple<rmm::device_uvector<int64_t>, rmm::device_uvector<int64_t>> lookup(
  raft::handle_t const& handle,
  graph_view_t<int64_t, int64_t, false, false> const& graph_view,
  edge_property_view_t<int64_t, int64_t const*> edge_id_view,
  edge_property_view_t<int64_t, uint8_t const*> edge_type_view,
  edge_type_and_id_search_container_t<uint8_t, int64_t, thrust::tuple<int64_t, int64_t>> const&
    search_container,
  raft::device_span<int64_t const> edge_ids_to_lookup,
  uint8_t edge_type_to_lookup);

}  // namespace cugraph
