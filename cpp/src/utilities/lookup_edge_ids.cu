/*
 * Copyright (c) 2021-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "detail/graph_partition_utils.cuh"

#include <cugraph/detail/shuffle_wrappers.hpp>
#include <cugraph/edge_partition_view.hpp>
#include <cugraph/graph.hpp>
#include <cugraph/graph_functions.hpp>
#include <cugraph/graph_view.hpp>
#include <cugraph/utilities/shuffle_comm.cuh>

#include <raft/comms/mpi_comms.hpp>
#include <raft/core/comms.hpp>
#include <raft/core/handle.hpp>
#include <raft/random/rng_state.hpp>

#include <thrust/for_each.h>
#include <thrust/tuple.h>

#include <iostream>
#include <string>
#include <tuple>

namespace cugraph {
namespace detail {

/**
 * @brief This function prints vertex and edge partitions.
 */
template <typename vertex_t, typename edge_t, bool store_transposed, bool multi_gpu>
std::
  tuple<rmm::device_uvector<edge_t>, rmm::device_uvector<vertex_t>, rmm::device_uvector<vertex_t>>
  lookup_edge_ids_impl(
    raft::handle_t const& handle,
    cugraph::graph_view_t<vertex_t, edge_t, store_transposed, multi_gpu> const& graph_view,
    std::optional<cugraph::edge_property_view_t<edge_t, edge_t const*>> edge_id_view,
    raft::device_span<edge_t const> edge_ids_to_lookup)
{
  auto const comm_rank = multi_gpu ? handle.get_comms().get_rank() : 0;
  bool has_edge_id     = false;
  if (edge_id_view.has_value()) { has_edge_id = true; }

  rmm::device_uvector<edge_t> sorted_edge_ids_to_lookup(edge_ids_to_lookup.size(),
                                                        handle.get_stream());

  raft::copy(sorted_edge_ids_to_lookup.begin(),
             edge_ids_to_lookup.begin(),
             edge_ids_to_lookup.size(),
             handle.get_stream());

  thrust::sort(
    handle.get_thrust_policy(), sorted_edge_ids_to_lookup.begin(), sorted_edge_ids_to_lookup.end());

  rmm::device_uvector<vertex_t> output_srcs(sorted_edge_ids_to_lookup.size(), handle.get_stream());
  rmm::device_uvector<vertex_t> output_dsts(sorted_edge_ids_to_lookup.size(), handle.get_stream());

  auto constexpr invalid_partner = cugraph::invalid_vertex_id<vertex_t>::value;
  thrust::fill(handle.get_thrust_policy(), output_srcs.begin(), output_srcs.end(), invalid_partner);
  thrust::fill(handle.get_thrust_policy(), output_dsts.begin(), output_dsts.end(), invalid_partner);

  rmm::device_uvector<edge_t> matched_edge_indices(sorted_edge_ids_to_lookup.size(),
                                                   handle.get_stream());

  thrust::fill(handle.get_thrust_policy(),
               matched_edge_indices.begin(),
               matched_edge_indices.end(),
               cugraph::invalid_vertex_id<edge_t>::value);

  //
  // Read sources and destinations associated with ege ids
  //

  for (size_t ep_idx = 0; ep_idx < graph_view.number_of_local_edge_partitions(); ++ep_idx) {
    auto edge_partition_view = graph_view.local_edge_partition_view(ep_idx);

    auto number_of_edges_in_edge_partition = edge_partition_view.number_of_edges();
    auto offsets                           = edge_partition_view.offsets();
    auto indices                           = edge_partition_view.indices();

    assert(number_of_edges_in_edge_partition == indices.size());

    auto major_range_first = edge_partition_view.major_range_first();
    auto major_range_last  = edge_partition_view.major_range_last();

    auto major_hypersparse_first = edge_partition_view.major_hypersparse_first();
    auto dcs_nzd_vertices        = edge_partition_view.dcs_nzd_vertices();

    raft::device_span<edge_t const> ids_of_edges_stored_in_this_edge_partition{};

    if (has_edge_id) {
      auto value_firsts = edge_id_view->value_firsts();
      auto edge_counts  = edge_id_view->edge_counts();

      ids_of_edges_stored_in_this_edge_partition =
        raft::device_span<edge_t const>(value_firsts[ep_idx], edge_counts[ep_idx]);
    }

    thrust::for_each(
      handle.get_thrust_policy(),
      thrust::make_counting_iterator(size_t{0}),
      thrust::make_counting_iterator(size_t{ids_of_edges_stored_in_this_edge_partition.size()}),
      [has_edge_id,
       indices,
       stored_edge_ids = ids_of_edges_stored_in_this_edge_partition.begin(),
       sorted_edge_ids_to_lookup =
         raft::device_span<edge_t const>{sorted_edge_ids_to_lookup.begin(),
                                         sorted_edge_ids_to_lookup.size()},

       matched_edge_indices = matched_edge_indices.begin()] __device__(auto edge_idx) {
        if (has_edge_id) {
          auto ptr = thrust::lower_bound(thrust::seq,
                                         sorted_edge_ids_to_lookup.begin(),
                                         sorted_edge_ids_to_lookup.end(),
                                         stored_edge_ids[edge_idx]);
          if (*ptr == stored_edge_ids[edge_idx]) {
            matched_edge_indices[ptr - sorted_edge_ids_to_lookup.begin()] = edge_idx;

            auto w_at = static_cast<std::ptrdiff_t>(ptr - sorted_edge_ids_to_lookup.begin());
            printf("\neidx = %d dst = %d at = %d\n",
                   static_cast<int>(edge_idx),
                   static_cast<int>(indices[edge_idx]),
                   static_cast<int>(w_at));
          }
        }
      });

    auto last_in_non_hypersparse =
      (major_hypersparse_first ? (*major_hypersparse_first) : major_range_last) - major_range_first;
    rmm::device_uvector<vertex_t> uppers(matched_edge_indices.size(), handle.get_stream());

    thrust::fill(handle.get_thrust_policy(),
                 uppers.begin(),
                 uppers.end(),
                 cugraph::invalid_vertex_id<vertex_t>::value);

    thrust::upper_bound(handle.get_thrust_policy(),
                        offsets.begin(),
                        offsets.end(),
                        matched_edge_indices.begin(),
                        matched_edge_indices.end(),
                        uppers.begin());

#if 1
    RAFT_CUDA_TRY(hipDeviceSynchronize());

    auto title = std::string("offsets_")
                   .append(std::to_string(comm_rank))
                   .append("_")
                   .append(std::to_string(ep_idx));

    raft::print_device_vector(title.c_str(), offsets.begin(), offsets.size(), std::cout);

    title = std::string("matched_")
              .append(std::to_string(comm_rank))
              .append("_")
              .append(std::to_string(ep_idx));

    raft::print_device_vector(
      title.c_str(), matched_edge_indices.begin(), matched_edge_indices.size(), std::cout);

    title = std::string("uppers_")
              .append(std::to_string(comm_rank))
              .append("_")
              .append(std::to_string(ep_idx));

    raft::print_device_vector(title.c_str(), uppers.begin(), uppers.size(), std::cout);

#endif

    bool is_hypersparse = major_hypersparse_first.has_value();

    thrust::transform(
      handle.get_thrust_policy(),
      thrust::make_zip_iterator(thrust::make_tuple(uppers.begin(), matched_edge_indices.begin())),
      thrust::make_zip_iterator(thrust::make_tuple(uppers.end(), matched_edge_indices.end())),
      thrust::make_zip_iterator(thrust::make_tuple(output_srcs.begin(), output_dsts.begin())),
      [major_range_first,
       indices,
       stored_edge_ids = ids_of_edges_stored_in_this_edge_partition.begin(),
       invalid_id      = cugraph::invalid_vertex_id<vertex_t>::value,
       is_hypersparse,
       major_hypersparse_first = (*major_hypersparse_first),
       dcs_nzd_vertices        = (*dcs_nzd_vertices)] __device__(auto upper_and_eidx) {
        vertex_t idx_in_offsets = thrust::get<0>(upper_and_eidx) - 1;
        auto eidx               = thrust::get<1>(upper_and_eidx);
        vertex_t major;

        if (is_hypersparse && (idx_in_offsets >= (major_hypersparse_first - major_range_first))) {
          major = dcs_nzd_vertices[idx_in_offsets];
        } else {
          major = idx_in_offsets;
        }
        if (eidx >= 0) {
          printf("> %d (%d) : %d %d\n",
                 static_cast<int>(stored_edge_ids[eidx]),
                 static_cast<int>(eidx),
                 static_cast<int>(major),
                 static_cast<int>(indices[eidx]));

          return thrust::make_tuple(major, indices[eidx]);

        } else {
          return thrust::make_tuple(invalid_id, invalid_id);
        }
      });

#if 1
    // Edge property values
    if (edge_id_view) {
      auto value_firsts = edge_id_view->value_firsts();
      auto edge_counts  = edge_id_view->edge_counts();

      assert(number_of_edges_in_edge_partition == edge_counts[ep_idx]);

      RAFT_CUDA_TRY(hipDeviceSynchronize());
      auto edge_ids_title = std::string("edge_ids_")
                              .append(std::to_string(comm_rank))
                              .append("_")
                              .append(std::to_string(ep_idx));
      raft::print_device_vector(
        edge_ids_title.c_str(), value_firsts[ep_idx], number_of_edges_in_edge_partition, std::cout);
    }
#endif
  }

  return std::make_tuple(
    std::move(sorted_edge_ids_to_lookup), std::move(output_srcs), std::move(output_dsts));
}

}  // namespace detail

template <typename vertex_t, typename edge_t, bool store_transposed, bool multi_gpu>
std::
  tuple<rmm::device_uvector<edge_t>, rmm::device_uvector<vertex_t>, rmm::device_uvector<vertex_t>>
  lookup_edge_ids(
    raft::handle_t const& handle,
    cugraph::graph_view_t<vertex_t, edge_t, store_transposed, multi_gpu> const& graph_view,
    std::optional<cugraph::edge_property_view_t<edge_t, edge_t const*>> edge_id_view,
    raft::device_span<edge_t const> edge_ids_to_lookup)
{
  return detail::lookup_edge_ids_impl(handle, graph_view, edge_id_view, edge_ids_to_lookup);
}

template std::
  tuple<rmm::device_uvector<int32_t>, rmm::device_uvector<int32_t>, rmm::device_uvector<int32_t>>
  lookup_edge_ids(raft::handle_t const& handle,
                  graph_view_t<int32_t, int32_t, false, true> const& graph_view,
                  std::optional<edge_property_view_t<int32_t, int32_t const*>> edge_id_view,
                  raft::device_span<int32_t const> edge_ids_to_lookup);

template std::
  tuple<rmm::device_uvector<int64_t>, rmm::device_uvector<int32_t>, rmm::device_uvector<int32_t>>
  lookup_edge_ids(raft::handle_t const& handle,
                  graph_view_t<int32_t, int64_t, false, true> const& graph_view,
                  std::optional<edge_property_view_t<int64_t, int64_t const*>> edge_id_view,
                  raft::device_span<int64_t const> edge_ids_to_lookup);

template std::
  tuple<rmm::device_uvector<int64_t>, rmm::device_uvector<int64_t>, rmm::device_uvector<int64_t>>
  lookup_edge_ids(raft::handle_t const& handle,
                  graph_view_t<int64_t, int64_t, false, true> const& graph_view,
                  std::optional<edge_property_view_t<int64_t, int64_t const*>> edge_id_view,
                  raft::device_span<int64_t const> edge_ids_to_lookup);

template std::
  tuple<rmm::device_uvector<int32_t>, rmm::device_uvector<int32_t>, rmm::device_uvector<int32_t>>
  lookup_edge_ids(raft::handle_t const& handle,
                  graph_view_t<int32_t, int32_t, false, false> const& graph_view,
                  std::optional<edge_property_view_t<int32_t, int32_t const*>> edge_id_view,
                  raft::device_span<int32_t const> edge_ids_to_lookup);

template std::
  tuple<rmm::device_uvector<int64_t>, rmm::device_uvector<int32_t>, rmm::device_uvector<int32_t>>
  lookup_edge_ids(raft::handle_t const& handle,
                  graph_view_t<int32_t, int64_t, false, false> const& graph_view,
                  std::optional<edge_property_view_t<int64_t, int64_t const*>> edge_id_view,
                  raft::device_span<int64_t const> edge_ids_to_lookup);

template std::
  tuple<rmm::device_uvector<int64_t>, rmm::device_uvector<int64_t>, rmm::device_uvector<int64_t>>
  lookup_edge_ids(raft::handle_t const& handle,
                  graph_view_t<int64_t, int64_t, false, false> const& graph_view,
                  std::optional<edge_property_view_t<int64_t, int64_t const*>> edge_id_view,
                  raft::device_span<int64_t const> edge_ids_to_lookup);

}  // namespace cugraph
